#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

/////////////////////////////////////////////////////////////////////////////////////////
/* Laufzeitvergleich von einer Datei zwischen 
    verschiedenen Auslastungsgraden einer Hashtabelle und
    verschiedenen offenen Hashverfahren bei

    a. einer gegebenen Anzahl von Schlüsseln, 
    b. gleichen oder unterschiedlichen Schlüsselgrößen und 
    c. einer gegebenen 1. und 2. Hashfunktionen
 */
/////////////////////////////////////////////////////////////////////////////////////////

template <typename T1, typename T2>
void runKernel(int block_num, int num_threads_per_block){
    int deviceID{0};
    struct hipDeviceProp_t props;
    const size_t matrix_size{block_num*num_threads_per_block * sizeof(cell<T1,T2>)};

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);
    
    std::cout << "GPU" << "," << props.name << std::endl;
    std::cout << "VRAM" << "," << (props.totalGlobalMem/1024)/1024 << "MB" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten" << ",";
    std::cout << ((matrix_size * 3 + sizeof(cell<T1,T2>)) / 1024 / 1024) << "MB\n" << std::endl;
    std::cout << "Block_Zahl" << "," << "Threads_Zahl_Pro_Block" << std::endl;
    std::cout << block_num << "," << num_threads_per_block << std::endl;
    std::cout << std::endl;     
};

//Führe Hashverfahren mit verschiedenen Datentypen aus
template <typename T1, typename T2>
void runMain(hash_type type, hash_function function1, hash_function function2, size_t key_num, size_t hash_table_size, double occupancy, char* fileName,
             int block_num = 0, int num_threads_per_block = 0){
    const size_t exampleOccupyKeyNum{(size_t) ceil((double) (hash_table_size) * occupancy)};
   
    std::cout << "Anzahl der gespeicherten Zellen" << "," << key_num << std::endl;
    if (type != cuckoo_probe){
        std::cout << "Größe der Hashtabelle" << "," << hash_table_size << std::endl;
    }else{
        std::cout << "Größe der Cuckoo-Hashtabelle" << "," << 2*hash_table_size << std::endl;
    }
    std::cout << "Auslastungsfaktor der Hashtabelle" << "," << occupancy << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<T1,T2> example_hash_table(key_num, exampleOccupyKeyNum, hash_table_size,function1,function2,
                                                 block_num, num_threads_per_block);
    example_hash_table.readCells(fileName);
    example_hash_table.insertTestCells2(type);
};

int main(int argc, char** argv){
    //1. Deklariere die Variablen
    char* fileName;
    int exampleBlockNum, exampleThreadsPerBlock;
    const double * occupancy = new double[6]{1.0,0.8,0.6,0.4,0.2,0.0};
    size_t exampleKeyNum, exampleHashTableSize;
    int function_code1, function_code2;
    hash_function hash_function1, hash_function2;
    
    if(argc < 8){
        std::cout << "Fehler bei der Eingabe von Parametern" << std::endl;
        return -1;
    }

    fileName = argv[1];
    exampleKeyNum = (size_t) atoi(argv[2]);
    exampleHashTableSize = (size_t) atoi(argv[3]);  
    function_code1 = atoi(argv[4]);
    function_code2 = atoi(argv[5]);
    exampleBlockNum = atoi(argv[6]);
    exampleThreadsPerBlock= atoi(argv[7]);

    if (exampleHashTableSize < 1){
        std::cout << "Die Größe einer oder zwei Hashtabelle(n) muss mehr als Null betragen." << std::endl;
        return -1;
    }

    if (function_code1<1 || function_code1>11){
        std::cout << "Der Code einer 1. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    if (function_code2<1 || function_code2>11){
        std::cout << "Der Code einer 2. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    if (exampleBlockNum < 1 || exampleThreadsPerBlock < 1 || 
        exampleKeyNum != (size_t)(exampleBlockNum*exampleThreadsPerBlock)){
        exampleBlockNum = (int) exampleKeyNum;
        exampleThreadsPerBlock = 1;
    }

    runKernel<uint32_t,uint32_t>(exampleBlockNum, exampleThreadsPerBlock);

    if (function_code1 == 2){
        hash_function1 = multiplication;
        std::cout << "1. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code1 == 3){
        hash_function1 = murmer;
        std::cout << "1. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code1 == 4){
        hash_function1 = universal0;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 5){
        hash_function1 = universal1;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 6){
        hash_function1 = universal2;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 7){
        hash_function1 = dycuckoo_hash1;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code1 == 8){
        hash_function1 = dycuckoo_hash2;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code1 == 9){
        hash_function1 = dycuckoo_hash3;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code1 == 10){
        hash_function1 = dycuckoo_hash4;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code1 == 11) {
        hash_function1 = dycuckoo_hash5;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function1 = modulo;
        std::cout << "1. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }

    if (function_code2 == 2){
        hash_function2 = multiplication;
        std::cout << "2. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code2 == 3){
        hash_function2 = murmer;
        std::cout << "2. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code2 == 4){
        hash_function2 = universal0;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 5){
        hash_function2 = universal1;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 6){
        hash_function2 = universal2;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 7){
        hash_function2 = dycuckoo_hash1;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code2 == 8){
        hash_function2 = dycuckoo_hash2;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code2 == 9){
        hash_function2 = dycuckoo_hash3;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code2 == 10){
        hash_function2 = dycuckoo_hash4;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code2 == 11) {
        hash_function2 = dycuckoo_hash5;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function2 = modulo;
        std::cout << "2. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }
    std::cout << std::endl;
    
    CPUTimer timer;
    timer.start();

    /////////////////////////////////////////////////////////////////////////////////////////
    //Lineare Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<6; i++) runMain<uint32_t,uint32_t>(linear_probe, hash_function1, hash_function2, exampleKeyNum, exampleHashTableSize,occupancy[i], fileName,
                                                            exampleBlockNum, exampleThreadsPerBlock);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Quadratische Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<6; i++) runMain<uint32_t,uint32_t>(quadratic_probe, hash_function1, hash_function2, exampleKeyNum, exampleHashTableSize,occupancy[i], fileName, 
                                                            exampleBlockNum, exampleThreadsPerBlock);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Doppelte Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<6; i++) runMain<uint32_t,uint32_t>(double_probe, hash_function1, hash_function2, exampleKeyNum, exampleHashTableSize, occupancy[i], fileName, 
                                                            exampleBlockNum, exampleThreadsPerBlock);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Cuckoo-Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<6; i++) runMain<uint32_t,uint32_t>(cuckoo_probe, hash_function1, hash_function2, exampleKeyNum, exampleHashTableSize, occupancy[i], fileName, 
                                                            exampleBlockNum, exampleThreadsPerBlock);
    /////////////////////////////////////////////////////////////////////////////////////////

    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer" << "," << timer.getDuration() << std::endl;
    
    return 0;
};