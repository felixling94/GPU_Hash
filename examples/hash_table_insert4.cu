#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

/////////////////////////////////////////////////////////////////////////////////////////
//Laufzeitvergleich von einer Datei zwischen verschiedenen Auslastungsgraden einer Hashtabelle bei
//a. einer gegebenen Anzahl von Schlüsseln, 
//b. gleichen oder unterschiedlichen Schlüsselgrößen, 
//c. einer gegebenen 1. und 2. Hashfunktionen, und
//d. gegebenen Hashverfahren, z.B. linearem Sondieren
/////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
void runKernel(int block_num, int num_threads_per_block){
    int deviceID{0};
    struct hipDeviceProp_t props;
    const size_t matrix_size{block_num*num_threads_per_block * sizeof(T)};

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);
    
    std::cout << "GPU" << "," << props.name << std::endl;
    std::cout << "VRAM" << "," << (props.totalGlobalMem/1024)/1024 << "MB" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten" << ",";
    std::cout << ((matrix_size * 3 + sizeof(uint32_t)) / 1024 / 1024) << "MB\n" << std::endl;
    std::cout << "Block_Zahl" << "," << "Threads_Zahl_Pro_Block" << std::endl;
    std::cout << block_num << "," << num_threads_per_block << std::endl;
    std::cout << std::endl;     
};

//Führe Hashverfahren mit verschiedenen Datentypen aus
template <typename T>
void runMain(hash_type type, hash_function function1, hash_function function2, int block_num, int num_threads_per_block, double occupancy, char* fileName){
    const size_t hashTableSize{(size_t) ceil((double) (block_num * num_threads_per_block) / occupancy)};
   
    std::cout << "Anzahl der gespeicherten Zellen" << "," << block_num*num_threads_per_block << std::endl;
    if (type != cuckoo_probe){
        std::cout << "Größe der Hashtabelle" << "," << hashTableSize << std::endl;
    }else{
        std::cout << "Größe der Cuckoo-Hashtabelle" << "," << 2*hashTableSize << std::endl;
    }
    std::cout << "Auslastungsfaktor der Hashtabelle" << "," << occupancy << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<T> example_hash_table(block_num, num_threads_per_block,hashTableSize,function1,function2);
    example_hash_table.readCells(fileName);
    example_hash_table.insertTestCells2(type);
};

int main(int argc, char** argv){
    //1. Deklariere die Variablen
    char* fileName;
    int exampleBlockNum, exampleThreadsPerBlock;
    const double * occupancy = new double[5]{1.0,0.8,0.6,0.4,0.2};

    size_t * exampleHashTableSize = new size_t[5];
    int function_code1, function_code2;
    hash_function hash_function1, hash_function2;
    
    if(argc < 6){
        std::cout << "Fehler bei der Eingabe von Parametern" << std::endl;
        return -1;
    }

    fileName = argv[1];
    exampleBlockNum = atoi(argv[2]);
    exampleThreadsPerBlock= atoi(argv[3]);
    function_code1 = atoi(argv[4]);
    function_code2 = atoi(argv[5]);

    if (exampleBlockNum <=0){
        std::cout << "Die Anzahl an Blöcke muss mehr als Null betragen." << std::endl;
        return -1;
    }

    if (exampleThreadsPerBlock <=0){
        std::cout << "Die Anzahl an Threads pro Block muss mehr als Null betragen." << std::endl;
        return -1;
    }

    if (function_code1<1 || function_code1>11){
        std::cout << "Der Code einer 1. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    if (function_code2<1 || function_code2>11){
        std::cout << "Der Code einer 2. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    runKernel<uint32_t>(exampleBlockNum, exampleThreadsPerBlock);

    if (function_code1 == 2){
        hash_function1 = multiplication;
        std::cout << "1. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code1 == 3){
        hash_function1 = murmer;
        std::cout << "1. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code1 == 4){
        hash_function1 = universal0;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 5){
        hash_function1 = universal1;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 6){
        hash_function1 = universal2;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 7){
        hash_function1 = dycuckoo_hash1;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code1 == 8){
        hash_function1 = dycuckoo_hash2;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code1 == 9){
        hash_function1 = dycuckoo_hash3;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code1 == 10){
        hash_function1 = dycuckoo_hash4;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code1 == 11) {
        hash_function1 = dycuckoo_hash5;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function1 = modulo;
        std::cout << "1. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }

    if (function_code2 == 2){
        hash_function2 = multiplication;
        std::cout << "2. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code2 == 3){
        hash_function2 = murmer;
        std::cout << "2. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code2 == 4){
        hash_function2 = universal0;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 5){
        hash_function2 = universal1;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 6){
        hash_function2 = universal2;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 7){
        hash_function2 = dycuckoo_hash1;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code2 == 8){
        hash_function2 = dycuckoo_hash2;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code2 == 9){
        hash_function2 = dycuckoo_hash3;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code2 == 10){
        hash_function2 = dycuckoo_hash4;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code2 == 11) {
        hash_function2 = dycuckoo_hash5;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function2 = modulo;
        std::cout << "2. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }
    std::cout << std::endl;
    
    CPUTimer timer;
    timer.start();

    /////////////////////////////////////////////////////////////////////////////////////////
    //Lineare Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<5; i++) runMain<uint32_t>(linear_probe, hash_function1, hash_function2, exampleBlockNum, exampleThreadsPerBlock, occupancy[i], fileName);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Quadratische Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<5; i++) runMain<uint32_t>(quadratic_probe, hash_function1, hash_function2, exampleBlockNum, exampleThreadsPerBlock, occupancy[i], fileName);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Doppelte Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<5; i++) runMain<uint32_t>(double_probe, hash_function1, hash_function2, exampleBlockNum, exampleThreadsPerBlock, occupancy[i], fileName);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Cuckoo-Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    for (size_t i = 0; i<5; i++) runMain<uint32_t>(cuckoo_probe, hash_function1, hash_function2, exampleBlockNum, exampleThreadsPerBlock, occupancy[i], fileName);
    /////////////////////////////////////////////////////////////////////////////////////////

    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer" << "," << timer.getDuration() << std::endl;
    
    return 0;
};