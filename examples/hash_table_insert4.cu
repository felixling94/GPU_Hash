#include <iostream>
#include <cmath>
#include <algorithm>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

/////////////////////////////////////////////////////////////////////////////////////////
//Laufzeitvergleich zwischen verschiedener Anzahl von Schlüsseln bei  
//a. einer gegebenen 1. und 2. Hashfunktionen, und
//b. gegebenen Hashverfahren, z.B. linearem Sondieren
/////////////////////////////////////////////////////////////////////////////////////////

//Führe Hashverfahren mit verschiedenen Datentypen aus
template <typename T1, typename T2>
void runMain(hash_type type, hash_function function1, hash_function function2, size_t keyNum){
    int deviceID{0};
    struct hipDeviceProp_t props;
    const size_t matrix_size{keyNum * sizeof(T1)};
    const size_t hashTableSize{(size_t) (keyNum*120/100)};

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);
    
    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;
    std::cout << "Ausgewähltes " << props.name << " mit "
              << (props.totalGlobalMem/1024)/1024 << "mb VRAM" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten: "
              << (( matrix_size * 3 + sizeof(T1)) / 1024 / 1024) << "mb\n" << std::endl;
    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;

    if (function1 == multiplication){
        std::cout << "1. Hashfunktion: Multiplikative Methode" << std::endl;
    }else if (function1 == murmer){
        std::cout << "1. Hashfunktion: Murmer Hash" << std::endl;
    }else if (function1 == universal0){
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 290000  b: 320000  Primzahl: 320114)" << std::endl;
    }else if (function1 == universal1){
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 149400  b: 149500  Primzahl: 149969)" << std::endl;
    }else if (function1 == universal2){
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 135  b: 140  Primzahl: 149)" << std::endl;
    }else if (function1 == dycuckoo_hash1){
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 1" << std::endl;
    }else if (function1 == dycuckoo_hash2){
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 2" << std::endl;
    }else if (function1 == dycuckoo_hash3){
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 3" << std::endl;
    }else if (function1 == dycuckoo_hash4){
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 4" << std::endl;
    }else if (function1 == dycuckoo_hash5) {
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 5" << std::endl;
    }else{
        std::cout << "1. Hashfunktion: Divisions-Rest-Methode" << std::endl;
    }

    if (function2 == multiplication){
        std::cout << "2. Hashfunktion: Multiplikative Methode" << std::endl;
    }else if (function2 == murmer){
        std::cout << "2. Hashfunktion: Murmer Hash" << std::endl;
    }else if (function2 == universal0){
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 290000  b: 320000  Primzahl: 320114)" << std::endl;
    }else if (function2 == universal1){
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 149400  b: 149500  Primzahl: 149969)" << std::endl;
    }else if (function2 == universal2){
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 135  b: 140  Primzahl: 149)" << std::endl;
    }else if (function2 == dycuckoo_hash1){
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 1" << std::endl;
    }else if (function2 == dycuckoo_hash2){
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 2" << std::endl;
    }else if (function2 == dycuckoo_hash3){
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 3" << std::endl;
    }else if (function2 == dycuckoo_hash4){
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 4" << std::endl;
    }else if (function2 == dycuckoo_hash5) {
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 5" << std::endl;
    }else{
        std::cout << "2. Hashfunktion: Divisions-Rest-Methode" << std::endl;
    }
    std::cout << std::endl;

    std::cout << "Anzahl der gespeicherten Zellen             : ";
    std::cout << keyNum << std::endl;
    if (type != cuckoo_probe){
        std::cout << "Größe der Hashtabelle                       : ";
        std::cout << hashTableSize << std::endl;
    }else{
        std::cout << "Größe der Cuckoo-Hashtabelle                : ";
        std::cout << 2*hashTableSize << std::endl;
    }
    std::cout << std::endl;

    Example_Hash_Table<T1,T2> example_hash_table(keyNum,hashTableSize,function1,function2);
    example_hash_table.createCells(1,(int)keyNum*2);
    example_hash_table.insertTestCells2(type);
};

int main(int argc, char** argv){
    //1. Deklariere die Variablen
    const size_t * key_num = new size_t[5]{320114,25000,200000,140000,100019};
    size_t * exampleHashTableSize = new size_t[5];
    int function_code1, function_code2, hash_type_code;
    hash_function hash_function1, hash_function2;
    hash_type hash_type1; 
    
    if(argc < 4){
        std::cout << "Fehler bei der Eingabe von Parametern" << std::endl;
        return -1;
    }

    hash_type_code = atoi(argv[1]);
    function_code1 = atoi(argv[2]);
    function_code2 = atoi(argv[3]);

    if (hash_type_code<0 || hash_type_code>3){
        std::cout << "Der Kode eines Hashtyps muss innerhalb des Bereiches von 0 bis 3 sein." << std::endl;
        return -1;
    }

    if (function_code1<1 || function_code1>11){
        std::cout << "Der Kode einer 1. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    if (function_code2<1 || function_code2>11){
        std::cout << "Der Kode einer 2. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }
    
    if (hash_type_code == 1){
        hash_type1 = quadratic_probe;
    }else if(hash_type_code == 2){
        hash_type1 = double_probe;
    }else if(hash_type_code == 3){
        hash_type1 = cuckoo_probe;
    }else{
        hash_type1 = linear_probe;
    }

    if (function_code1 == 2){
        hash_function1 = multiplication;
    }else if (function_code1 == 3){
        hash_function1 = murmer;
    }else if (function_code1 == 4){
        hash_function1 = universal0;
    }else if (function_code1 == 5){
        hash_function1 = universal1;
    }else if (function_code1 == 6){
        hash_function1 = universal2;
    }else if (function_code1 == 7){
        hash_function1 = dycuckoo_hash1;
    }else if (function_code1 == 8){
        hash_function1 = dycuckoo_hash2;
    }else if (function_code1 == 9){
        hash_function1 = dycuckoo_hash3;
    }else if (function_code1 == 10){
        hash_function1 = dycuckoo_hash4;
    }else if (function_code1 == 11) {
        hash_function1 = dycuckoo_hash5;
    }else{
        hash_function1 = modulo;
    }

    if (function_code2 == 2){
        hash_function2 = multiplication;
    }else if (function_code2 == 3){
        hash_function2 = murmer;
    }else if (function_code2 == 4){
        hash_function2 = universal0;
    }else if (function_code2 == 5){
        hash_function2 = universal1;
    }else if (function_code2 == 6){
        hash_function2 = universal2;
    }else if (function_code2 == 7){
        hash_function2 = dycuckoo_hash1;
    }else if (function_code2 == 8){
        hash_function2 = dycuckoo_hash2;
    }else if (function_code2 == 9){
        hash_function2 = dycuckoo_hash3;
    }else if (function_code2 == 10){
        hash_function2 = dycuckoo_hash4;
    }else if (function_code2 == 11) {
        hash_function2 = dycuckoo_hash5;
    }else{
        hash_function2 = modulo;
    }

    CPUTimer timer;
    timer.start();

    for (size_t i = 0; i<5; i++) runMain<uint32_t,uint32_t>(hash_type1, hash_function1, hash_function2, key_num[i]);
  
    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer                                 : ";
    std::cout << timer.getDuration() << std::endl;
    
    return 0;
};