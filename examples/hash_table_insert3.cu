#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

/////////////////////////////////////////////////////////////////////////////////////////
//Laufzeitvergleich zwischen verschiedenen Auslastungsgraden einer Hashtabelle bei
//a. einer gegebenen Anzahl von Schlüsseln, 
//b. gleichen oder unterschiedlichen Schlüsselgrößen, 
//c. einer gegebenen 1. und 2. Hashfunktionen, und
//d. gegebenen Hashverfahren, z.B. linearem Sondieren
/////////////////////////////////////////////////////////////////////////////////////////

const size_t key_num{576};

template <typename T>
void runKernel(){
    int deviceID{0};
    struct hipDeviceProp_t props;
    const size_t matrix_size{key_num * sizeof(T)};

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);
    
    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;
    std::cout << "Ausgewähltes " << props.name << " mit "
              << (props.totalGlobalMem/1024)/1024 << "mb VRAM" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten: "
              << (( matrix_size * 3 + sizeof(T)) / 1024 / 1024) << "mb\n" << std::endl;

};

//Führe Hashverfahren mit verschiedenen Datentypen aus
template <typename T>
void runMain(hash_type type, hash_function function1, hash_function function2, double occupancy, bool key_length_same){
    const size_t hashTableSize{(size_t) ceil((double) (key_num) / occupancy)};
   
    std::cout << "Anzahl der gespeicherten Zellen             : ";
    std::cout << key_num << std::endl;
    if (type != cuckoo_probe){
        std::cout << "Größe der Hashtabelle                       : ";
        std::cout << hashTableSize << std::endl;
    }else{
        std::cout << "Größe der Cuckoo-Hashtabelle                : ";
        std::cout << 2*hashTableSize << std::endl;
    }
    std::cout << "Auslastungsfaktor der Hashtabelle           : ";
    std::cout << occupancy << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<T> example_hash_table(key_num,hashTableSize,function1,function2);
    example_hash_table.createCells(key_length_same);
    example_hash_table.insertTestCells2(type);
    
    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;
};

int main(int argc, char** argv){
    //1. Deklariere die Variablen
    const double * occupancy = new double[5]{1.0,0.8,0.6,0.4,0.2};
    size_t * exampleHashTableSize = new size_t[5];
    int function_code1, function_code2, hash_type_code, int_key_length_same;
    hash_function hash_function1, hash_function2;
    hash_type hash_type1;
    bool key_length_same; 
    
    if(argc < 5){
        std::cout << "Fehler bei der Eingabe von Parametern" << std::endl;
        return -1;
    }

    int_key_length_same = atoi(argv[1]);
    hash_type_code = atoi(argv[2]);
    function_code1 = atoi(argv[3]);
    function_code2 = atoi(argv[4]);

    if (int_key_length_same<0 || int_key_length_same>1){
        std::cout << "Der Kode der Gleichheit der Schlüsselgröße muss entweder 0 bis 1 sein." << std::endl;
        return -1;
    }

    if (hash_type_code<0 || hash_type_code>3){
        std::cout << "Der Kode eines Hashtyps muss innerhalb des Bereiches von 0 bis 3 sein." << std::endl;
        return -1;
    }

    if (function_code1<1 || function_code1>11){
        std::cout << "Der Kode einer 1. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }

    if (function_code2<1 || function_code2>11){
        std::cout << "Der Kode einer 2. Hashfunktion muss innerhalb des Bereiches von 1 bis 11 sein." << std::endl;
        return -1;
    }
    
    if (hash_type_code == 1){
        hash_type1 = quadratic_probe;
    }else if(hash_type_code == 2){
        hash_type1 = double_probe;
    }else if(hash_type_code == 3){
        hash_type1 = cuckoo_probe;
    }else{
        hash_type1 = linear_probe;
    }

    runKernel<uint32_t>();
    
    if (int_key_length_same == 1){
        key_length_same = true;
    }else{
        key_length_same = false;     
    }
      
    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;
    if (function_code1 == 2){
        hash_function1 = multiplication;
        std::cout << "1. Hashfunktion: Multiplikative Methode" << std::endl;
    }else if (function_code1 == 3){
        hash_function1 = murmer;
        std::cout << "1. Hashfunktion: Murmer Hash" << std::endl;
    }else if (function_code1 == 4){
        hash_function1 = universal0;
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 290000  b: 320000  Primzahl: 320114)" << std::endl;
    }else if (function_code1 == 5){
        hash_function1 = universal1;
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 149400  b: 149500  Primzahl: 149969)" << std::endl;
    }else if (function_code1 == 6){
        hash_function1 = universal2;
        std::cout << "1. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 135  b: 140  Primzahl: 149)" << std::endl;
    }else if (function_code1 == 7){
        hash_function1 = dycuckoo_hash1;
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 1" << std::endl;
    }else if (function_code1 == 8){
        hash_function1 = dycuckoo_hash2;
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 2" << std::endl;
    }else if (function_code1 == 9){
        hash_function1 = dycuckoo_hash3;
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 3" << std::endl;
    }else if (function_code1 == 10){
        hash_function1 = dycuckoo_hash4;
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 4" << std::endl;
    }else if (function_code1 == 11) {
        hash_function1 = dycuckoo_hash5;
        std::cout << "1. Hashfunktion: DyCuckoo-Hash 5" << std::endl;
    }else{
        hash_function1 = modulo;
        std::cout << "1. Hashfunktion: Divisions-Rest-Methode" << std::endl;
    }

    if (function_code2 == 2){
        hash_function2 = multiplication;
        std::cout << "2. Hashfunktion: Multiplikative Methode" << std::endl;
    }else if (function_code2 == 3){
        hash_function2 = murmer;
        std::cout << "2. Hashfunktion: Murmer Hash" << std::endl;
    }else if (function_code2 == 4){
        hash_function2 = universal0;
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 290000  b: 320000  Primzahl: 320114)" << std::endl;
    }else if (function_code2 == 5){
        hash_function2 = universal1;
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 149400  b: 149500  Primzahl: 149969)" << std::endl;
    }else if (function_code2 == 6){
        hash_function2 = universal2;
        std::cout << "2. Hashfunktion: Universelle Hashfunktion" << std::endl;
        std::cout << "                 (a: 135  b: 140  Primzahl: 149)" << std::endl;
    }else if (function_code2 == 7){
        hash_function2 = dycuckoo_hash1;
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 1" << std::endl;
    }else if (function_code2 == 8){
        hash_function2 = dycuckoo_hash2;
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 2" << std::endl;
    }else if (function_code2 == 9){
        hash_function2 = dycuckoo_hash3;
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 3" << std::endl;
    }else if (function_code2 == 10){
        hash_function2 = dycuckoo_hash4;
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 4" << std::endl;
    }else if (function_code2 == 11) {
        hash_function2 = dycuckoo_hash5;
        std::cout << "2. Hashfunktion: DyCuckoo-Hash 5" << std::endl;
    }else{
        hash_function2 = modulo;
        std::cout << "2. Hashfunktion: Divisions-Rest-Methode" << std::endl;
    }
    std::cout << std::endl;
    
    CPUTimer timer;
    timer.start();
    
    for (size_t i = 0; i<5; i++) runMain<uint32_t>(hash_type1, hash_function1, hash_function2, occupancy[i],key_length_same);
  
    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer                                 : ";
    std::cout << timer.getDuration() << std::endl;
    
    return 0;
};