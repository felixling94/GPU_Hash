#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <random>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

int main(int argc, char** argv){
    //1. Deklariere die Variablen
    size_t exampleHashTableSize, exampleKeyNum, matrix_size;
    double occupancy;
    int function_code1, function_code2, int_key_length_same;
    hash_function hash_function1, hash_function2;
    bool key_length_same; 

    int deviceID{0};
    struct hipDeviceProp_t props;

    if(argc < 6){
        std::cout << "Fehler bei der Eingabe von Parametern" << std::endl;
        return -1;
    }

    exampleKeyNum = (size_t) atoi(argv[1]);
    int_key_length_same = atoi(argv[2]);
    occupancy = atof(argv[3]);
    function_code1 = atoi(argv[4]);
    function_code2 = atoi(argv[5]);
    
    if (exampleKeyNum <=0){
        std::cout << "Die Größe einer Schlüssel muss mehr als Null betragen." << std::endl;
        return -1;
    }

    if (int_key_length_same<0 || int_key_length_same>1){
        std::cout << "Der Code der Gleichheit der Schlüsselgröße muss entweder 0 bis 1 sein." << std::endl;
        return -1;
    }

    if (occupancy <=0){
        std::cout << "Der Auslastungsfaktor der Hashtabelle muss mehr als Null betragen." << std::endl;
        return -1;
    }
    
    if (function_code1<0 || function_code1>12){
        std::cout << "Der Code einer 1. Hashfunktion muss innerhalb des Bereiches von 0 bis 12 sein." << std::endl;
        return -1;
    }

    if (function_code2<0 || function_code2>12){
        std::cout << "Der Code einer 2. Hashfunktion muss innerhalb des Bereiches von 0 bis 12 sein." << std::endl;
        return -1;
    }

    matrix_size = exampleKeyNum * sizeof(uint32_t);
    exampleHashTableSize = (size_t) ceil((double) (exampleKeyNum) / occupancy);

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);

    std::cout << "GPU" << "," << props.name << std::endl;
    std::cout << "VRAM" << "," << (props.totalGlobalMem/1024)/1024 << "MB" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten" << ",";
    std::cout << ((matrix_size * 3 + sizeof(uint32_t)) / 1024 / 1024) << "MB\n" << std::endl;
    std::cout << std::endl;
    std::cout << "Anzahl der gespeicherten Zellen" << "," << exampleKeyNum << std::endl;
    std::cout << "Größe der Hashtabelle" << "," << exampleHashTableSize << std::endl;
    std::cout << "Größe der Cuckoo-Hashtabellen" << "," << 2*exampleHashTableSize << std::endl;
    std::cout << std::endl;

    if (int_key_length_same == 1){
        key_length_same = true;
    }else{
        key_length_same = false;     
    }

    if (function_code1 == 2){
        hash_function1 = multiplication;
        std::cout << "1. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code1 == 3){
        hash_function1 = murmer;
        std::cout << "1. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code1 == 4){
        hash_function1 = universal0;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 5){
        hash_function1 = universal1;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 6){
        hash_function1 = universal2;
        std::cout << "1. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code1 == 7){
        hash_function1 = dycuckoo_hash1;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code1 == 8){
        hash_function1 = dycuckoo_hash2;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code1 == 9){
        hash_function1 = dycuckoo_hash3;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code1 == 10){
        hash_function1 = dycuckoo_hash4;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code1 == 11) {
        hash_function1 = dycuckoo_hash5;
        std::cout << "1. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function1 = modulo;
        std::cout << "1. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }

    if (function_code2 == 2){
        hash_function2 = multiplication;
        std::cout << "2. Hashfunktion" << "," << "Multiplikative Methode" << std::endl;
    }else if (function_code2 == 3){
        hash_function2 = murmer;
        std::cout << "2. Hashfunktion" << "," << "Murmer-Hashfunktion" << std::endl;
    }else if (function_code2 == 4){
        hash_function2 = universal0;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 20019" << "," << "b: 20025" << "," <<  "Primzahl: 20029" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 5){
        hash_function2 = universal1;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 10023" << "," << "b: 10037" << "," <<  "Primzahl: 10039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 6){
        hash_function2 = universal2;
        std::cout << "2. Hashfunktion" << "," << "Universelle Hashfunktion" << std::endl;
        std::cout << "," << "a: 5029" << "," << "b: 5038" << "," <<  "Primzahl: 5039" << std::endl;
        std::cout << std::endl;
    }else if (function_code2 == 7){
        hash_function2 = dycuckoo_hash1;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-1" << std::endl;
    }else if (function_code2 == 8){
        hash_function2 = dycuckoo_hash2;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-2" << std::endl;
    }else if (function_code2 == 9){
        hash_function2 = dycuckoo_hash3;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-3" << std::endl;
    }else if (function_code2 == 10){
        hash_function2 = dycuckoo_hash4;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-4" << std::endl;
    }else if (function_code2 == 11) {
        hash_function2 = dycuckoo_hash5;
        std::cout << "2. Hashfunktion" << "," << "DyCuckoo-5" << std::endl;
    }else{
        hash_function2 = modulo;
        std::cout << "2. Hashfunktion" << "," << "Divisions-Rest-Methode" << std::endl;
    }

    std::cout << std::endl;
    Example_Hash_Table<uint32_t> example_hash_table(exampleKeyNum,exampleHashTableSize,hash_function1,hash_function2);
    example_hash_table.createCells(key_length_same);

    CPUTimer timer;
    timer.start();

    /////////////////////////////////////////////////////////////////////////////////////////
    //Keine Kollionsauflösung
    /////////////////////////////////////////////////////////////////////////////////////////
    example_hash_table.searchTestCells2(no_probe);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Lineare Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    example_hash_table.searchTestCells2(linear_probe);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Quadratische Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    example_hash_table.searchTestCells2(quadratic_probe);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Doppelte Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    example_hash_table.searchTestCells2(double_probe);
    /////////////////////////////////////////////////////////////////////////////////////////
    //Cuckoo-Hashverfahren
    /////////////////////////////////////////////////////////////////////////////////////////
    example_hash_table.searchTestCells2(cuckoo_probe);
    /////////////////////////////////////////////////////////////////////////////////////////

    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer" << "," << timer.getDuration() << std::endl;
    
    return 0;
};