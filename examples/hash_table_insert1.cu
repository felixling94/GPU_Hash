#include <iostream>
#include <cmath>
#include <algorithm>
#include <vector>
#include <random>
#include <stdint.h>

#include "example_hash_table.cuh"
#include <../include/base.h>
#include <../tools/timer.cuh>

/////////////////////////////////////////////////////////////////////////////////////////
//Leistungsvergleich zwischen verschiedenen Hashfunktionen
/////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv){
    //1. Deklariere die Variablen
    const size_t exampleKeyNum{90*60*60};
    const size_t exampleHashTableSize{exampleKeyNum};
    const size_t matrix_size{exampleKeyNum * sizeof(uint32_t)};

    int deviceID{0};
    struct hipDeviceProp_t props;

    hipSetDevice(deviceID);
	hipGetDeviceProperties(&props, deviceID);

    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;
    std::cout << "Ausgewähltes " << props.name << " mit "
              << (props.totalGlobalMem/1024)/1024 << "mb VRAM" << std::endl;
    std::cout << "Gesamtgröße von Kernelargumenten: "
              << (( matrix_size * 3 + sizeof(uint32_t)) / 1024 / 1024) << "mb\n" << std::endl;

    std::cout << "****************************************************************";
    std::cout << "***************" << std::endl;   
    std::cout << "Anzahl der gespeicherten Zellen             : ";
    std::cout << exampleKeyNum << std::endl;
    std::cout << "Größe der Hashtabelle                       : ";
    std::cout << exampleHashTableSize << std::endl;

    CPUTimer timer;
    timer.start();

    /////////////////////////////////////////////////////////////////////////////////////////
    //Modulo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "1. Hashfunktion: Divisions-Rest-Methode" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table1(exampleKeyNum,exampleHashTableSize,modulo);
    example_hash_table1.createCells(1,(int)exampleKeyNum*2);
    example_hash_table1.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //Multiplikative Methode
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "2. Hashfunktion: Multiplikative Methode" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table2(exampleKeyNum,exampleHashTableSize,multiplication);
    example_hash_table2.createCells(1,(int)exampleKeyNum*2);
    example_hash_table2.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //Murmer-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "3. Hashfunktion: Murmer-Hashfunktion" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table3(exampleKeyNum,exampleHashTableSize,murmer);
    example_hash_table3.createCells(1,(int)exampleKeyNum*2);
    example_hash_table3.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //1. Universelle Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "4. Hashfunktion: Universelle Hashfunktion" << std::endl;
    std::cout << "                 (a: 290000  b: 320000  Primzahl: 320114)" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table4(exampleKeyNum,exampleHashTableSize,universal0);
    example_hash_table4.createCells(1,(int)exampleKeyNum*2);
    example_hash_table4.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //2. Universelle Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "5. Hashfunktion: Universelle Hashfunktion" << std::endl;
    std::cout << "                 (a: 149400  b: 149500  Primzahl: 149969)" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table5(exampleKeyNum,exampleHashTableSize,universal1);
    example_hash_table5.createCells(1,(int)exampleKeyNum*2);
    example_hash_table5.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //3. Universelle Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "6. Hashfunktion: Universelle Hashfunktion" << std::endl;
    std::cout << "                 (a: 135  b: 140  Primzahl: 149)" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table6(exampleKeyNum,exampleHashTableSize,universal2);
    example_hash_table6.createCells(1,(int)exampleKeyNum*2);
    example_hash_table6.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //1. DyCuckoo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "7. Hashfunktion: DyCuckoo-Hash 1" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table7(exampleKeyNum,exampleHashTableSize,dycuckoo_hash1);
    example_hash_table7.createCells(1,(int)exampleKeyNum*2);
    example_hash_table7.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //2. DyCuckoo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "9. Hashfunktion: DyCuckoo-Hash 2" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table8(exampleKeyNum,exampleHashTableSize,dycuckoo_hash2);
    example_hash_table8.createCells(1,(int)exampleKeyNum*2);
    example_hash_table8.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //3. DyCuckoo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "9. Hashfunktion: DyCuckoo-Hash 3" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table9(exampleKeyNum,exampleHashTableSize,dycuckoo_hash3);
    example_hash_table9.createCells(1,(int)exampleKeyNum*2);
    example_hash_table9.insertTestCells2(no_probe);


    /////////////////////////////////////////////////////////////////////////////////////////
    //4. DyCuckoo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "10. Hashfunktion: DyCuckoo-Hash 4" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table10(exampleKeyNum,exampleHashTableSize,dycuckoo_hash4);
    example_hash_table10.createCells(1,(int)exampleKeyNum*2);
    example_hash_table10.insertTestCells2(no_probe);

    /////////////////////////////////////////////////////////////////////////////////////////
    //5. DyCuckoo-Hashfunktion
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << std::endl;
    std::cout << "11. Hashfunktion: DyCuckoo-Hash 5" << std::endl;
    std::cout << std::endl;

    Example_Hash_Table<uint32_t,uint32_t> example_hash_table11(exampleKeyNum,exampleHashTableSize,dycuckoo_hash5);
    example_hash_table11.createCells(1,(int)exampleKeyNum*2);
    example_hash_table11.insertTestCells2(no_probe);

    //Fasse Resultate zusammen
    timer.stop();
    std::cout << std::endl;
    std::cout << "Gesamtdauer für alle Hashfunktionen ohne    : ";
    std::cout << timer.getDuration() << std::endl;
    std::cout << "Kollionsauflösung (in Mikrosekunden)" << std::endl;
    
    return 0;
};